


#include <hip/hip_runtime.h>
#include <bits/stdc++.h>


int SumReduction(const int *input, int n) {
  int sum = 0;
  for (int i = 0; i < n; ++i) {
    sum += input[i];
  }
  return sum;
}


__global__ void SimpleSumReductionKernel(int *input, int n, int *output) {
  unsigned int i = 2 * threadIdx.x;
  for (int stride = 1; stride <= blockDim.x; stride *= 2) {
    if (threadIdx.x % stride == 0 && i + stride < n) {
      input[i] += input[i + stride];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    *output = input[0];
  }
}


void TestSimpleSumReductionKernel() {
  int n = 1024;
  std::vector<int> h_inp(n, 1);
  std::iota(h_inp.begin(), h_inp.end(), 0);
  int h_out{12};

  int *d_inp, *d_out;
  hipMalloc(&d_inp, sizeof(int) * n);
  hipMemcpy(d_inp, h_inp.data(), n * sizeof(int), hipMemcpyHostToDevice);
  hipMalloc(&d_out, sizeof(int));

  SimpleSumReductionKernel<<<1, n / 2>>>(d_inp, n, d_out);
  int gt = SumReduction(h_inp.data(), n);
  
  hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_inp);
  hipFree(d_out);
  
  hipDeviceSynchronize();

  std::cout << "SumReduction: " << h_out << ' ' << gt << '\n';
  assert(h_out == gt);
}


int main() {
  TestSimpleSumReductionKernel();
}
